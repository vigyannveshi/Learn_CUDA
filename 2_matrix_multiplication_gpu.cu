#include "hip/hip_runtime.h"
/* 
Matrix Multiplication using GPU
--> This is not a optimized GPU program
--> It can be further accelerated using several primitives that GPU offer to it.
 */
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <hip/hip_runtime.h>

__global__
void MatrixMulKernel(float * d_M, float* d_N, float* d_P, int n){
    int i=blockIdx.y*blockDim.y+threadIdx.y;
    int j=blockIdx.x*blockDim.x+threadIdx.x;
    if ((i<n) && (j<n)){
        float Pvalue =0.0;
        for(int k=0; k<n;k++){
            Pvalue+=d_M[i*n+k]*d_N[k*n+j];
        }
        d_P[i*n+j]=Pvalue;
    }
}

int main() {
    int n = 500;
    int size = n * n;

    // Allocate memory for host matrices
    float *h_M = (float *)malloc(size * sizeof(float));
    float *h_N = (float *)malloc(size * sizeof(float));
    float *h_P = (float *)malloc(size * sizeof(float));

    // Initialize host matrices with some values
    for (int i = 0; i < size; i++) {
        h_M[i] = i;  // Example initialization, you can set your own values
        h_N[i] = i;
    }

    // Allocate memory on the device for matrices
    float *d_M, *d_N, *d_P;
    hipMalloc(&d_M, size * sizeof(float));
    hipMalloc(&d_N, size * sizeof(float));
    hipMalloc(&d_P, size * sizeof(float));

    // Transfer host matrices to device
    hipMemcpy(d_M, h_M, size * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(d_N, h_N, size * sizeof(float), hipMemcpyHostToDevice);

    // Define grid and block dimensions
    dim3 grid(2, 2);    // 2x2 grid
    dim3 block(8, 8);   // 8x8 threads per block

    // Perform matrix multiplication on GPU
    MatrixMulKernel<<<grid, block>>>(d_M, d_N, d_P, n);

    // Transfer result matrix from device to host
    hipMemcpy(h_P, d_P, size * sizeof(float), hipMemcpyDeviceToHost);

    // Verify result (printing only a part of the matrix for brevity)
    // printf("M: ");
    // for (int i = 0; i < n; i++) {
    //     for (int j = 0; j < n; j++) {
    //         printf("%.2f\t", h_M[i * n + j]);
    //     }
    //     printf("\n");
    // }
    // printf("N: ");
    // for (int i = 0; i < n; i++) {
    //     for (int j = 0; j < n; j++) {
    //         printf("%.2f\t", h_N[i * n + j]);
    //     }
    //     printf("\n");
    // }
    // printf("Result matrix (partial):\n");
    // for (int i = 0; i < n; i++) {
    //     for (int j = 0; j < n; j++) {
    //         printf("%.2f\t", h_P[i * n + j]);
    //     }
    //     printf("\n");
    // }

    printf("successfully calculated!");

    
    // Free host and device memory
    free(h_M);
    free(h_N);
    free(h_P);
    hipFree(d_M);
    hipFree(d_N);
    hipFree(d_P);

    return 0;
}