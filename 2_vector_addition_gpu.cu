// header files
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>



// CPU side host program
void vecAdd(float* h_A, float *h_B, float *h_C, int n){
    int size = n*sizeof(float);
    float *d_A=NULL, *d_B=NULL, *d_C=NULL;

    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    // Initiating vectors on cuda device
    printf("Initiate device vectors\n");
    err=hipMalloc((void **)&d_A, size);
    if (err !=hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n",hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err=hipMalloc((void **)&d_B, size);
    if (err !=hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector B (error code %s)!\n",hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err=hipMalloc((void **)&d_C, size);
    if (err !=hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n",hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }


    // Copying vectors from host memory to the CUDA device
    printf("Copy input data from host memory to the CUDA device\n");

    err=hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

    if (err !=hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n",hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err=hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    if (err !=hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector B from host to device (error code %s)!\n",hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err=hipMemcpy(d_C, h_C, size, hipMemcpyHostToDevice);

    if (err !=hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector C from host to device (error code %s)!\n",hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    int threads    

}
