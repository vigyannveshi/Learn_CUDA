/* 
Vector Addition GPU:
--> to compile the code:
    `nvcc kernel.cu host.cu -o vectorAdd`
--> to run the code:
    `./vectorAdd`
*/

// header files
#include <hip/hip_runtime.h>


/* Kernel function Prototype*/
__global__ void vectorAdd(float *, float *,float *, int);

/* Kernel function Implementation/Definition */
__global__
void vectorAdd(float *A, float *B, float *C, int n ){
    int i=threadIdx.x+blockDim.x*blockIdx.x;
    if (i<n)
        C[i]=A[i]+B[i];
}


/* CPU side host program */ 
void vecAdd(float* h_A, float *h_B, float *h_C, int n){
    int size = n*sizeof(float);
    float *d_A=NULL, *d_B=NULL, *d_C=NULL;

    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    // Initiating vectors on cuda device
    printf("Initiate device vectors\n");
    err=hipMalloc((void **)&d_A, size);
    if (err !=hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n",hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err=hipMalloc((void **)&d_B, size);
    if (err !=hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector B (error code %s)!\n",hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err=hipMalloc((void **)&d_C, size);
    if (err !=hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n",hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }


    // Copying vectors from host memory to the CUDA device
    printf("Copy input data from host memory to the CUDA device\n");

    err=hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

    if (err !=hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n",hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err=hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    if (err !=hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector B from host to device (error code %s)!\n",hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err=hipMemcpy(d_C, h_C, size, hipMemcpyHostToDevice);

    if (err !=hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector C from host to device (error code %s)!\n",hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Launching Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (n+threadsPerBlock -1 )/threadsPerBlock;
    printf("CUDA kernel launch with %d blocks of %d threads \n", threadsPerBlock,blocksPerGrid);

    /* GPU KERNEL */
    vectorAdd<<<blocksPerGrid,threadsPerBlock>>>(d_A, d_B, d_C, n);
    /* GPU KERNEL */

    err=hipGetLastError();

    // device function (CUDA kernel) called from host does not have return type
    // CUDA runtime functions (execute in host side) can have return type

    if (err !=hipSuccess)
    {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n",hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    // Device to Host Memory Transfer
    printf("Copy output data from output device to the host memory\n");
    err = hipMemcpy(h_C ,d_C ,size , hipMemcpyDeviceToHost);
    if (err !=hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n",hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Free all the allocated memory in the GPU
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Verify that the result vector is correct 
    for (int i = 0; i<n; i++){
        if (fabs(h_A[i]+h_B[i]-h_C[i])>1e-5){
            fprintf(stderr, "Result verification failed at element %d\n",i);
            exit(EXIT_FAILURE);
        }
    }
    printf("Test Passed");
}
