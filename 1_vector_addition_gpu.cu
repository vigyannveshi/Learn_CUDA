#include "hip/hip_runtime.h"
/* 
Vector Addition GPU:
*/

// header files
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

/* Kernel function Prototype*/
// __global__ void vectorAdd(float *, float *,float *, int);

/* Kernel function Implementation/Definition */
__global__ void vectorAdd(const float *A,const  float *B, float *C, int n ){
    int i=threadIdx.x+blockDim.x*blockIdx.x;
    if (i<n){
        C[i]=A[i]+B[i];
    }
}


/* CPU side host program */ 
void vecAdd(float* h_A, float *h_B, float *h_C, int n){
    int size = n*sizeof(float);
    float *d_A=NULL, *d_B=NULL, *d_C=NULL;

    // Error code to check return values for CUDA calls
    hipError_t err = hipSuccess;

    // Initiating vectors on cuda device
    printf("Initiate device vectors\n");
    err=hipMalloc((void **)&d_A, size);
    if (err !=hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector A (error code %s)!\n",hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err=hipMalloc((void **)&d_B, size);
    if (err !=hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector B (error code %s)!\n",hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    err=hipMalloc((void **)&d_C, size);
    if (err !=hipSuccess)
    {
        fprintf(stderr, "Failed to allocate device vector C (error code %s)!\n",hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }


    // Copying vectors from host memory to the CUDA device
    printf("Copy input data from host memory to the CUDA device\n");

    err=hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

    if (err !=hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector A from host to device (error code %s)!\n",hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err=hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);

    if (err !=hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector B from host to device (error code %s)!\n",hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    err=hipMemcpy(d_C, h_C, size, hipMemcpyHostToDevice);

    if (err !=hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector C from host to device (error code %s)!\n",hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Launching Kernel
    int threadsPerBlock = 256;
    int blocksPerGrid = (n+threadsPerBlock -1 )/threadsPerBlock;
    printf("CUDA kernel launch with %d blocks of %d threads \n", threadsPerBlock,blocksPerGrid);

    /* GPU KERNEL */
    vectorAdd<<<blocksPerGrid,threadsPerBlock>>>(d_A, d_B, d_C, n);
    /* GPU KERNEL */

    err=hipGetLastError();

    // device function (CUDA kernel) called from host does not have return type
    // CUDA runtime functions (execute in host side) can have return type

    if (err !=hipSuccess)
    {
        fprintf(stderr, "Failed to launch vectorAdd kernel (error code %s)!\n",hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }
    
    // Device to Host Memory Transfer
    printf("Copy output data from output device to the host memory\n");
    err = hipMemcpy(h_C ,d_C ,size , hipMemcpyDeviceToHost);
    if (err !=hipSuccess)
    {
        fprintf(stderr, "Failed to copy vector C from device to host (error code %s)!\n",hipGetErrorString(err));
        exit(EXIT_FAILURE);
    }

    // Free all the allocated memory in the GPU
    hipFree(d_A);
    hipFree(d_B);
    hipFree(d_C);

    // Verify that the result vector is correct 
    for (int i = 0; i<n; i++){
        if (fabs(h_A[i]+h_B[i]-h_C[i])>1e-5){
            fprintf(stderr, "Result verification failed at element %d\n",i);
            exit(EXIT_FAILURE);
        }
    }
    printf("Test Passed");
}

int main(void){
    int n;
    printf("Enter the vector length: ");
    scanf("%d",&n);
    int size = n*sizeof(float);

    // Allocate the host input vector A
    float *h_A = (float *)malloc(size);

    // Allocate the host input vector B
    float *h_B = (float *)malloc(size);

    // Allocate the host output vector C
    float *h_C = (float *)malloc(size);

    // Verify that allocations succeeded
    if (h_A == NULL || h_B == NULL || h_C == NULL)
    {
        fprintf(stderr, "Failed to allocate host vectors!\n");
        exit(EXIT_FAILURE);
    }

    // Initialize the host input vectors
    for (int i = 0; i < n; ++i)
    {
        h_A[i] = rand()/(float)RAND_MAX;
        h_B[i] = rand()/(float)RAND_MAX;
    }

    // Running the vector addition

    vecAdd(h_A, h_B, h_C, n);
    return 0;
}